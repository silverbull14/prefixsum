#include "hip/hip_runtime.h"
#include <iostream>
#include<stdio.h>
#include<stdlib.h>
#include<cstdio>
#include<time.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
using namespace std;
void generate(int* a, int size, int max, int min) {
	for (unsigned long long int i = 0; size > i; i++) {
		a[i] = rand() % max + 1 + min;
	}
}
__global__ void prefix_sum(int* in, int* out, int tn, int bn) {
	int  tid = threadIdx.x, bid = blockIdx.x, sid = blockIdx.x * tn + tid, p = 1, q = 0;
	//printf("in[%d]:%d\n ", sid, in[sid]);
	//printf("\n");
	extern __shared__ int add[];
	add[tid] = in[sid];
	for (int z = 1; tn > z; z *= 2) {
		p = 1 - p; //0 1 0 1
		q = 1 - p; //1 0 1 0
		if (tid >= z) {
			add[q * tn + tid] = add[p * tn + tid] + add[p * tn + tid - z];
			//printf("add[%d]:%d\n ", sid, add[q * tn + tid]);
			//printf("\n");
		}
		else {
			add[q * tn + tid] = add[p * tn + tid];
			//printf("add[%d]:%d\n ", sid, add[q * tn + tid]);
			//printf("\n");
		}
		__syncthreads();
	}
	//printf("out[%d]:%d\n ", sid, out[sid]);
	//printf("\n");
	out[sid] = add[q * tn + tid];
	out[sid] += out[bid * tn - 1];
	__syncthreads();
}

bool test(int a[], int b[], int max) {
	return 0;
}
__global__ void cpuprefixsum(int* array, int data) {
	for (int i = 1; data > i; i++) {
		array[i] = array[i - 1] + array[i];
	}
}
main() {
	srand(time(NULL));
	while (1) {
		int data; int maxr = 10, minr = 0, n, tn, ctr[1] = { 1 };
		clock_t start_t, stop_t;
		double time;
		hipEvent_t start, stop;
		float htd, dth, kernel, cpukernel;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		printf("輸入要連加的整數數量(2的n次方):");
		cin >> n;
		data = pow(2, n);
		tn = (data < 1024) ? data : 1024;
		//計算幾格block
		int bn = (data % tn == 0) ? data / tn : data / tn + 1;
		//獲取陣列
		int* array = new int[data];
		//產生亂數
		generate(array, data, maxr, minr);
		//顯示初始陣列
		/*for (int i = 0; data > i; i++) {
			printf(" %d", array[i]);
		}*/
		//printf("\n");
		//獲取gpu陣列
		int* gpuarray, * result = new int[data], * cpukernelarray;
		hipMalloc((void**)&gpuarray, data * sizeof(int));
		hipMalloc((void**)&cpukernelarray, data * sizeof(int));
		hipMemcpy(cpukernelarray, array, sizeof(int) * data, hipMemcpyHostToDevice);
		//CPU作法丟GPU跑
		hipEventRecord(start, 0);
		//cpuprefixsum << <1, 1 >> > (cpukernelarray, data);
		hipEventRecord(stop, 0); hipEventSynchronize(stop); hipEventElapsedTime(&cpukernel, start, stop);
		//將cpu陣列轉至gpu陣列
		hipEventRecord(start, 0);
		hipMemcpy(gpuarray, array, sizeof(int) * data, hipMemcpyHostToDevice);

		hipEventRecord(stop, 0); hipEventSynchronize(stop); hipEventElapsedTime(&dth, start, stop);
		//printf("x:%d", x);
		hipEventRecord(start, 0);
		prefix_sum << <bn, tn, tn * 2 * sizeof(int) >> > (gpuarray, gpuarray, tn, bn);
		hipEventRecord(stop, 0); hipEventSynchronize(stop); hipEventElapsedTime(&kernel, start, stop);
		hipEventRecord(start, 0);
		hipMemcpy(result, gpuarray, sizeof(int) * data, hipMemcpyDeviceToHost);
		hipEventRecord(stop, 0); hipEventSynchronize(stop); hipEventElapsedTime(&htd, start, stop);
		//CPU prefix_sum
		start_t = clock();
		for (int i = 1; data > i; i++) {
			array[i] = array[i - 1] + array[i];
		}
		stop_t = clock();
		time = double(stop_t - start_t) / CLOCKS_PER_SEC;
		//檢驗
		if (test(array, result, data)) {
			printf("incorrect\n");
		}
		else {
			printf("correct\n");
		}
		printf("CPU: %f\n", time);
		printf("CPU to GPU: %f\n", htd / 1000);
		printf("GPU: %f\n", kernel / 1000);
		printf("GPU to CPU: %f\n", dth / 1000);
		//printf("CPU in kernel: %f\n", cpukernel / 1000);
		//輸出result
		/*for (int i = 0; data > i; i++) {
			printf("[%d] %d\n", i, result[i]);
		}*/

	}
}
